// kernel.cu
#include <hip/hip_runtime.h>
#include <iostream>

__global__ void addKernel(int* c, const int* a, const int* b, int size) {
    int i = threadIdx.x;
    if (i < size) {
        c[i] = a[i] + b[i];
    }
}

void addWithCuda(int* c, const int* a, const int* b, int size) {
    int *dev_a, *dev_b, *dev_c;

    hipMalloc((void**)&dev_a, size * sizeof(int));
    hipMalloc((void**)&dev_b, size * sizeof(int));
    hipMalloc((void**)&dev_c, size * sizeof(int));

    hipMemcpy(dev_a, a, size * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(dev_b, b, size * sizeof(int), hipMemcpyHostToDevice);

    addKernel<<<1, size>>>(dev_c, dev_a, dev_b, size);

    hipMemcpy(c, dev_c, size * sizeof(int), hipMemcpyDeviceToHost);

    hipFree(dev_a);
    hipFree(dev_b);
    hipFree(dev_c);
}

